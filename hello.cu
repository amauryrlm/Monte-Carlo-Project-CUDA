#include "hip/hip_runtime.h"
#include <iostream>
// #include <format>
// #include <functional>
#include <hip/hip_runtime.h>

#include "trajectories.hpp"
#include "common.hpp"
#include "Xoshiro.hpp"
#include  "pricinghost.hpp"
#include <random>
#include <hiprand.h>


#include <math.h>
using namespace std;
// Function that catches the error
void testCUDA(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("There is an error in file %s at line %d\n", file, line);
        exit(EXIT_FAILURE);
    }
}
// Has to be defined in the compilation in order to get the correct value of the
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__, __LINE__))

///////////////////////////////////////////////////////////////////////////////
// Polynomial approximation of cumulative normal distribution function
///////////////////////////////////////////////////////////////////////////////
static double CND(double d)
{
    const double       A1 = 0.31938153;
    const double       A2 = -0.356563782;
    const double       A3 = 1.781477937;
    const double       A4 = -1.821255978;
    const double       A5 = 1.330274429;
    const double RSQRT2PI = 0.39894228040143267793994605993438;

    double
    K = 1.0 / (1.0 + 0.2316419 * fabs(d));

    double
    cnd = RSQRT2PI * exp(- 0.5 * d * d) *
          (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));

    if (d > 0)
        cnd = 1.0 - cnd;

    return cnd;
}


///////////////////////////////////////////////////////////////////////////////
// Black-Scholes formula for both call and put
///////////////////////////////////////////////////////////////////////////////
static void BlackScholesBodyCPU(
    float &callResult,
    float &putResult,
    float Sf, //Stock price
    float Xf, //Option strike
    float Tf, //Option years
    float Rf, //Riskless rate
    float Vf  //Volatility rate
)
{
    double S = Sf, X = Xf, T = Tf, R = Rf, V = Vf;

    double sqrtT = sqrt(T);
    double    d1 = (log(S / X) + (R + 0.5 * V * V) * T) / (V * sqrtT);
    double    d2 = d1 - V * sqrtT;
    double CNDD1 = CND(d1);
    double CNDD2 = CND(d2);

    //Calculate Call and Put simultaneously
    double expRT = exp(- R * T);
    callResult   = (float)(S * CNDD1 - X * expRT * CNDD2);
    putResult    = (float)(X * expRT * (1.0 - CNDD2) - S * (1.0 - CNDD1));
}



__device__ inline float cndGPU(float d)
{
    const float       A1 = 0.31938153f;
    const float       A2 = -0.356563782f;
    const float       A3 = 1.781477937f;
    const float       A4 = -1.821255978f;
    const float       A5 = 1.330274429f;
    const float RSQRT2PI = 0.39894228040143267793994605993438f;

    float
    K = __fdividef(1.0f, (1.0f + 0.2316419f * fabsf(d)));

    float
    cnd = RSQRT2PI * __expf(- 0.5f * d * d) *
          (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));

    if (d > 0)
        cnd = 1.0f - cnd;

    return cnd;
}


///////////////////////////////////////////////////////////////////////////////
// Black-Scholes formula for both call and put
///////////////////////////////////////////////////////////////////////////////
__device__ inline void BlackScholesBodyGPU(
    float &CallResult,
    float &PutResult,
    float S, //Stock price
    float X, //Option strike
    float T, //Option years
    float R, //Riskless rate
    float V  //Volatility rate
)
{
    float sqrtT, expRT;
    float d1, d2, CNDD1, CNDD2;

    sqrtT = __fdividef(1.0F, rsqrtf(T));
    d1 = __fdividef(__logf(S / X) + (R + 0.5f * V * V) * T, V * sqrtT);
    d2 = d1 - V * sqrtT;

    CNDD1 = cndGPU(d1);
    CNDD2 = cndGPU(d2);

    //Calculate Call and Put simultaneously
    expRT = __expf(- R * T);
    CallResult = S * CNDD1 - X * expRT * CNDD2;
    PutResult  = X * expRT * (1.0f - CNDD2) - S * (1.0f - CNDD1);
}


////////////////////////////////////////////////////////////////////////////////
//Process an array of optN options on GPU
////////////////////////////////////////////////////////////////////////////////
__launch_bounds__(128)
__global__ void BlackScholesGPU(
    float2 * __restrict d_CallResult,
    float2 * __restrict d_PutResult,
    float2 * __restrict d_StockPrice,
    float2 * __restrict d_OptionStrike,
    float2 * __restrict d_OptionYears,
    float Riskfree,
    float Volatility,
    int optN
)
{
    ////Thread index
    //const int      tid = blockDim.x * blockIdx.x + threadIdx.x;
    ////Total number of threads in execution grid
    //const int THREAD_N = blockDim.x * gridDim.x;

    const int opt = blockDim.x * blockIdx.x + threadIdx.x;

     // Calculating 2 options per thread to increase ILP (instruction level parallelism)
    if (opt < (optN / 2))
    {
        float callResult1, callResult2;
        float putResult1, putResult2;
        BlackScholesBodyGPU(
            callResult1,
            putResult1,
            d_StockPrice[opt].x,
            d_OptionStrike[opt].x,
            d_OptionYears[opt].x,
            Riskfree,
            Volatility
        );
        BlackScholesBodyGPU(
            callResult2,
            putResult2,
            d_StockPrice[opt].y,
            d_OptionStrike[opt].y,
            d_OptionYears[opt].y,
            Riskfree,
            Volatility
        );
        d_CallResult[opt] = make_float2(callResult1, callResult2);
        d_PutResult[opt] = make_float2(putResult1, putResult2);
	 }
}




void generateRandomArray(float *d_randomData, float *h_randomData, int N_PATHS, int N_STEPS, unsigned long long seed = 1234ULL){

    // create generator all fill array with generated values
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hiprandGenerateNormal(gen, d_randomData, N_PATHS * N_STEPS, 0.0, 1.0);
    cout << endl << "number generated" << endl;
    testCUDA(hipMemcpy(h_randomData, d_randomData, N_PATHS * N_STEPS * sizeof(float), hipMemcpyDeviceToHost));
    cout << "host copied" << endl;
    hiprandDestroyGenerator(gen);

}














__global__ void simulateOptionPriceGPU(float *d_optionPriceGPU, float K, float r, float T,float sigma, int N_PATHS, float *d_randomData, int N_STEPS, float S0, float dt, float sqrdt) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N_PATHS) {
        float St = S0;
        float G;
        for(int i = 0; i < N_STEPS; i++){
            G = d_randomData[idx*i];
            // cout << "G : " << G << endl;
            St *= exp((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
        }
        
        // // Calculate the payoff
        d_optionPriceGPU[idx] = max(St - K, 0.0f);


    }
}

//for one block

__global__ void simulateOptionPriceGPUSumReduce(float *d_optionPriceGPU, float K, float r, float T,float sigma, int N_PATHS, float *d_randomData, int N_STEPS, float S0, float dt, float sqrdt, float *output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    
    if (idx < N_PATHS) {
        float St = S0;
        float G;
        for(int i = 0; i < N_STEPS; i++){
            G = d_randomData[idx*i];
            // cout << "G : " << G << endl;
            St *= exp((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
        }

        St = max(St - K,0.0f);

    // Shared memory for the block
    __shared__ float sdata[1024];

    // Load input into shared memory
    sdata[tid] = (idx < N_PATHS) ? St : 0;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = N_PATHS / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to output
    if (tid == 0){
        output[0] = sdata[0];
        }
        
    } 
        
        
}

__global__ void simulateOptionPriceOneBlockGPUSumReduce(float *d_optionPriceGPU, float K, float r, float T,float sigma, int N_PATHS, float *d_randomData, int N_STEPS, float S0, float dt, float sqrdt, float *output) {
    int stride = blockDim.x;
    int idx = threadIdx.x;
    int tid = threadIdx.x;

    // Shared memory for the block
    __shared__ float sdata[1024];
    float sum = 0.0f;
    
    if(idx < N_PATHS) {
        sdata[idx] = 0.0f;

        while(idx < N_PATHS){
            float St = S0;
            float G;
            for(int i = 0; i < N_STEPS; i++){
                G = d_randomData[idx*i];
                // cout << "G : " << G << endl;
                St *= exp((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
            }
            sdata[tid] += max(St - K,0.0f);
            idx += stride;
        }
        



    // Load input into shared memory
        
        __syncthreads();

        // Perform reduction in shared memory
        for (unsigned int s = N_PATHS / 2; s > 0; s >>= 1) {
            if (tid < s) {
                sdata[tid] += sdata[tid + s];
            }
            __syncthreads();
        }

        // Write result for this block to output
        if (tid == 0){
            output[0] = sdata[0];
            }
            
        } 
        
        
}
void getDeviceProperty(){

    int count;
    hipDeviceProp_t prop;
    hipGetDeviceCount(&count);
    printf("The number of devices available is %i GPUs \n", count);
    hipGetDeviceProperties(&prop, count-1);
    printf("Name: %s\n", prop.name);
    printf("Global memory size in bytes: %ld\n", prop.totalGlobalMem);
    printf("Shared memory size per block: %ld\n", prop.sharedMemPerBlock);
    printf("Number of registers per block: %d\n", prop.regsPerBlock);
    printf("Number of threads in a warp: %d\n", prop.warpSize);
    printf("Maximum number of threads that can be launched per block: %d\n", prop.maxThreadsPerBlock);
    printf("Maximum number of threads that can be launched: %d x %d x %d\n",
           prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Maximum grid size: %d x %d x %d\n",
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Total constant memory: %ld\n", prop.totalConstMem);
    printf("Major compute capability: %d\n", prop.major);
    printf("Minor compute capability: %d\n", prop.minor);
    printf("Clock rate: %d\n", prop.clockRate);
    printf("Maximum 1D texture memory: %d\n", prop.maxTexture1D);
    printf("Could we overlap? %d\n", prop.deviceOverlap);
    printf("Number of multiprocessors: %d\n", prop.multiProcessorCount);
    printf("Is there a limit for kernel execution? %d\n", prop.kernelExecTimeoutEnabled);
    printf("Is my GPU a chipsest? %d\n", prop.integrated);
    printf("Can we map the host memory? %d\n", prop.canMapHostMemory);
    printf("Can we launch concurrent kernels? %d\n", prop.concurrentKernels);
    printf("Do we have ECC memory? %d\n", prop.ECCEnabled);

}

void simulateOptionPriceCPU(float *optionPriceCPU, int N_PATHS, int N_STEPS, float * h_randomData, float S0, float sigma, float sqrdt, float r, float K, float dt){
    float G;
    float countt = 0.0f;
    for(int i=0; i<N_PATHS;i++){
        float St = S0;
        for(int j=0; j<N_STEPS; j++){
            G = h_randomData[i*j];
            St *= exp((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
            
        }
        countt += max(St - K, 0.0f);
    }
    *optionPriceCPU = countt/N_PATHS;
}


int main(void) {



// declare variables and constants
    const size_t N_PATHS = 100000;
    const size_t N_STEPS = 1000;
    const size_t N_NORMALS = N_PATHS*N_STEPS;
    const float T = 1.0f;
    const float K = 100.0f;
    const float B = 95.0f;
    const float S0 = 100.0f;
    const float sigma = 0.2f;
    const float mu = 0.1f;
    const float r = 0.05f;
    float dt = float(T)/float(N_STEPS);
    float sqrdt = sqrt(dt);
    vector<float> s(N_PATHS);

    getDeviceProperty();


    float *d_randomData, *h_randomData;
    testCUDA(hipMalloc(&d_randomData, N_PATHS * N_STEPS * sizeof(float)));
    h_randomData = (float *)malloc(N_PATHS * N_STEPS*sizeof(float));
    generateRandomArray(d_randomData, h_randomData, N_PATHS, N_STEPS);


    cout << "random  " << h_randomData[0] << endl;

    float optionPriceCPU = 0.0f;
    simulateOptionPriceCPU(&optionPriceCPU,  N_PATHS,  N_STEPS,  h_randomData,  S0,  sigma,  sqrdt,  r, K, dt);

    cout << endl;

    cout << "Average CPU : " << optionPriceCPU << endl << endl;


    float *h_optionPriceGPU, *output;
    h_optionPriceGPU = (float *)malloc(N_PATHS * sizeof(float));
    output = (float *)malloc(sizeof(float));
    float *d_optionPriceGPU, *d_output;

    testCUDA(hipMalloc((void **)&d_optionPriceGPU,N_PATHS*sizeof(float)));
    testCUDA(hipMalloc((void **)&d_output,sizeof(float)));

    // simulateOptionPriceGPU<<<1, N_PATHS>>>( d_optionPriceGPU,  K,  r,  T, sigma,  N_PATHS,  d_randomData,  N_STEPS, S0, dt, sqrdt);
    // simulateOptionPriceGPUSumReduce<<<1, N_PATHS>>>( d_optionPriceGPU,  K,  r,  T, sigma,  N_PATHS,  d_randomData,  N_STEPS, S0, dt, sqrdt, d_output);
    simulateOptionPriceOneBlockGPUSumReduce<<<1, N_PATHS>>>( d_optionPriceGPU,  K,  r,  T, sigma,  N_PATHS,  d_randomData,  N_STEPS, S0, dt, sqrdt, d_output);

    hipDeviceSynchronize();


    hipMemcpy(h_optionPriceGPU, d_optionPriceGPU, N_PATHS * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(output, d_output, sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    cout << endl;

    cout << "Average GPU " << output[0]/ N_PATHS << endl ;
    float callResult = 0.0f;
    float putResult = 0.0f;

    BlackScholesBodyCPU(callResult,putResult,S0, K, T, r,  sigma);
    
    cout << "call BS" << callResult << endl;



    testCUDA(hipFree(d_randomData));


	return 0;
}


