#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include <hiprand.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <stdio.h>
#include <math.h>
#include "BlackandScholes.hpp"
#include "reduce.cuh"
#include "tool.cuh"
#include "option_price.hpp"

using namespace std;




// Function that catches the error
void testCUDA(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("There is an error in file %s at line %d\n", file, line);
        exit(EXIT_FAILURE);
    }
}
// Has to be defined in the compilation in order to get the correct value of the
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__, __LINE__))




__device__ inline float cndGPU(float d)
{
    const float       A1 = 0.31938153f;
    const float       A2 = -0.356563782f;
    const float       A3 = 1.781477937f;
    const float       A4 = -1.821255978f;
    const float       A5 = 1.330274429f;
    const float RSQRT2PI = 0.39894228040143267793994605993438f;

    float
    K = __fdividef(1.0f, (1.0f + 0.2316419f * fabsf(d)));

    float
    cnd = RSQRT2PI * __expf(- 0.5f * d * d) *
          (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));

    if (d > 0)
        cnd = 1.0f - cnd;

    return cnd;
}


///////////////////////////////////////////////////////////////////////////////
// Black-Scholes formula for both call and put
///////////////////////////////////////////////////////////////////////////////
__device__ inline void BlackScholesBodyGPU(
    float &CallResult,
    float &PutResult,
    float S, //Stock price
    float X, //Option strike
    float T, //Option years
    float R, //Riskless rate
    float V  //Volatility rate
)
{
    float sqrtT, expRT;
    float d1, d2, CNDD1, CNDD2;

    sqrtT = __fdividef(1.0F, rsqrtf(T));
    d1 = __fdividef(__logf(S / X) + (R + 0.5f * V * V) * T, V * sqrtT);
    d2 = d1 - V * sqrtT;

    CNDD1 = cndGPU(d1);
    CNDD2 = cndGPU(d2);

    //Calculate Call and Put simultaneously
    expRT = __expf(- R * T);
    CallResult = S * CNDD1 - X * expRT * CNDD2;
    PutResult  = X * expRT * (1.0f - CNDD2) - S * (1.0f - CNDD1);
}


////////////////////////////////////////////////////////////////////////////////
//Process an array of optN options on GPU
////////////////////////////////////////////////////////////////////////////////
__launch_bounds__(128)
__global__ void BlackScholesGPU(
    float2 * __restrict d_CallResult,
    float2 * __restrict d_PutResult,
    float2 * __restrict d_StockPrice,
    float2 * __restrict d_OptionStrike,
    float2 * __restrict d_OptionYears,
    float Riskfree,
    float Volatility,
    int optN
)
{
    ////Thread index
    //const int      tid = blockDim.x * blockIdx.x + threadIdx.x;
    ////Total number of threads in execution grid
    //const int THREAD_N = blockDim.x * gridDim.x;

    const int opt = blockDim.x * blockIdx.x + threadIdx.x;

     // Calculating 2 options per thread to increase ILP (instruction level parallelism)
    if (opt < (optN / 2))
    {
        float callResult1, callResult2;
        float putResult1, putResult2;
        BlackScholesBodyGPU(
            callResult1,
            putResult1,
            d_StockPrice[opt].x,
            d_OptionStrike[opt].x,
            d_OptionYears[opt].x,
            Riskfree,
            Volatility
        );
        BlackScholesBodyGPU(
            callResult2,
            putResult2,
            d_StockPrice[opt].y,
            d_OptionStrike[opt].y,
            d_OptionYears[opt].y,
            Riskfree,
            Volatility
        );
        d_CallResult[opt] = make_float2(callResult1, callResult2);
        d_PutResult[opt] = make_float2(putResult1, putResult2);
	 }
}




void generateRandomArray(float *d_randomData, float *h_randomData, int N_PATHS, int N_STEPS, unsigned long long seed = 1234ULL){

    // create generator all fill array with generated values
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hiprandGenerateNormal(gen, d_randomData, N_PATHS * N_STEPS, 0.0, 1.0);
    cout << endl << "number generated" << endl;
    testCUDA(hipMemcpy(h_randomData, d_randomData, N_PATHS * N_STEPS * sizeof(float), hipMemcpyDeviceToHost));
    cout << "host copied" << endl;
    hiprandDestroyGenerator(gen);

}






__global__ void simulateOptionPriceGPU(float *d_optionPriceGPU, float K, float r, float T,float sigma, int N_PATHS, float *d_randomData, int N_STEPS, float S0, float dt, float sqrdt) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N_PATHS) {
        float St = S0;
        float G;
        for(int i = 0; i < N_STEPS; i++){
            G = d_randomData[idx*N_STEPS + i];
            // cout << "G : " << G << endl;
            St *= exp((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
        }
        
        // // Calculate the payoff
        d_optionPriceGPU[idx] = max(St - K, 0.0f);


    }
}


__global__ void simulateOptionPriceOneBlockGPUSumReduce(float *d_optionPriceGPU, float K, float r, float T,float sigma, int N_PATHS, float *d_randomData, int N_STEPS, float S0, float dt, float sqrdt, float *output) {
    int stride = blockDim.x;
    int idx = threadIdx.x;
    int tid = threadIdx.x;

    // Shared memory for the block
    __shared__ float sdata[1024];
    float sum = 0.0f;
    
    if(idx < N_PATHS) {
        sdata[tid] = 0.0f;

        while(idx < N_PATHS){
            float St = S0;
            float G;
            for(int i = 0; i < N_STEPS; i++){
                G = d_randomData[idx*N_STEPS + i];
                // cout << "G : " << G << endl;
                St *= exp((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
            }
            sum += max(St - K,0.0f);
            idx += stride;
        }
    // Load input into shared memory
        sdata[tid] = (tid < N_PATHS) ? sum : 0;

        __syncthreads();

        // Perform reduction in shared memory
        for (unsigned int s = 1024 / 2; s > 0; s >>= 1) {
            if (tid < s && (tid + s) < N_PATHS) {
                sdata[tid] += sdata[tid + s];
            }
            __syncthreads();
        }

        // Write result for this block to output
        if (tid == 0){
            output[0] = sdata[0] * expf(-r);
            }  
        }  
}


__global__ void simulateOptionPriceMultipleBlockGPU(float *d_simulated_payoff, float K, float r, float T,float sigma, int N_PATHS, float *d_randomData, int N_STEPS, float S0, float dt, float sqrdt) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx < N_PATHS) {
            float St = S0;
            float G;
            for(int i = 0; i < N_STEPS; i++){
                G = d_randomData[idx*N_STEPS + i];
                St *= expf((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
            }
            d_simulated_payoff[idx] = max(St - K,0.0f);
        }
    }

__global__ void simulateBulletOptionPriceMultipleBlockGPU(float *d_simulated_payoff, float K, float r, float T,float sigma, int N_PATHS, float *d_randomData, int N_STEPS, float S0, float dt, float sqrdt, float B, float P1, float P2) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(idx < N_PATHS) {
    int count = 0;
    float St = S0;
    float G;
    for(int i = 0; i < N_STEPS; i++){
        G = d_randomData[idx*N_STEPS + i];
        St *= expf((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
        if(B > St) count +=1;
        }
    if((count >= P1) && (count <= P2)){
      d_simulated_payoff[idx] = max(St - K,0.0f);
    } else {
      d_simulated_payoff[idx] = 0.0f;
    }
  }
}



__global__ void simulateBulletOptionSavePrice(float *d_simulated_paths,float *d_simulated_count, float K, float r, float T,float sigma, int N_PATHS, float *d_randomData, int N_STEPS, float S0, float dt, float sqrdt, float B, float P1, float P2) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(idx < N_PATHS) {
    int count = 0;
    float St = S0;
    float G;
    for(int i = 0; i < N_STEPS; i++){
        G = d_randomData[idx*N_STEPS + i];
        St *= expf((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
        if(B > St) count +=1;
        if((count >= P1) && (count <= P2)){
          d_simulated_paths[idx * N_STEPS + i] = St;
        } else {
          d_simulated_paths[idx * N_STEPS + i] = 0.0f;
        }
        d_simulated_count[idx * N_STEPS + i] = count;
  }
}
}






int main(void) {



// declare variables and constants
    unsigned int N_PATHS = 1000000;
    const size_t N_STEPS = 100;
    const float T = 1.0f;
    const float K = 155.0;
    const float B = 110.0f;
    const float S0 = 156.30f;
    const float sigma = 0.2657f;
    const float r =  0.0237f;
    float dt = float(T)/float(N_STEPS);
    float sqrdt = sqrt(dt);
    int threadsPerBlock = 1024;
    unsigned int maxThreads = 1024;
    int P1 = 10;
    int P2 = 50;

    int block_sizes [6] = {32, 64, 128, 256, 512, 1024};
    int number_of_simulations [6] = {1000, 10000, 100000, 1000000, 10000000, 100000000};
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float times_for_simulations [6];

    FILE *file = fopen("simulation_results.csv", "w");
    fprintf(file, "number of simulations, 1000, 10000, 100000, 1000000, 10000000, 100000000\n");




    // getDeviceProperty();

    int blocksPerGrid = (N_PATHS + threadsPerBlock - 1) / threadsPerBlock;

    cout << "number of paths : " << N_PATHS << endl;
    cout << "number of steps : " << N_STEPS << endl;


    float *d_randomData, *h_randomData, *simulated_paths_cpu;
    testCUDA(hipMalloc(&d_randomData, N_PATHS * N_STEPS * sizeof(float)));
    h_randomData = (float *)malloc(N_PATHS * N_STEPS*sizeof(float));
    simulated_paths_cpu = (float *)malloc(N_PATHS *sizeof(float));
    generateRandomArray(d_randomData, h_randomData, N_PATHS, N_STEPS);


    cout << "random  " << h_randomData[0] << endl;

    float optionPriceCPU = 0.0f;
    simulateOptionPriceCPU(&optionPriceCPU,  N_PATHS,  N_STEPS,  h_randomData,  S0,  sigma,  sqrdt,  r, K, dt,simulated_paths_cpu);

    cout << endl;
    cout << "Average CPU : " << optionPriceCPU << endl << endl;



//--------------------------------GPU WITH ONE BLOCK ----------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------

    float *h_optionPriceGPU, *output;
    h_optionPriceGPU = (float *)malloc(N_PATHS * sizeof(float));
    output = (float *)malloc(sizeof(float));
    float *d_optionPriceGPU, *d_output;

    testCUDA(hipMalloc((void **)&d_optionPriceGPU,N_PATHS*sizeof(float)));
    testCUDA(hipMalloc((void **)&d_output,sizeof(float)));

    simulateOptionPriceOneBlockGPUSumReduce<<<1, 1024>>>( d_optionPriceGPU,  K,  r,  T, sigma,  N_PATHS,  d_randomData,  N_STEPS, S0, dt, sqrdt, d_output);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return -1;
    }
    hipMemcpy(h_optionPriceGPU, d_optionPriceGPU, N_PATHS * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(output, d_output, sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    cout << endl;

    cout << "Average GPU one block : " << output[0]/N_PATHS << endl ;

    hipFree(d_optionPriceGPU);
    hipFree(d_output);
    free(h_optionPriceGPU);
    free(output);

    int threads = 1024;
    int blocks = (N_PATHS + (threads * 2 - 1)) / (threads * 2);
//--------------------------------GPU WITH MULTIPLE BLOCK ----------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------
    float milliseconds = 0.0f;
    for(int i = 0; i < 6; i++){
      for(int j = 0; j < 6; j++){

        threads = block_sizes[i];
        N_PATHS = number_of_simulations[j];
        blocks = (N_PATHS + (threads * 2 - 1)) / (threads * 2);
        blocksPerGrid = (N_PATHS + threads - 1) / threads


        float *output3, *d_optionPriceGPU3, *d_output3;
        output3 = (float *)malloc(blocks * sizeof(float));

        testCUDA(hipMalloc((void **)&d_optionPriceGPU3,N_PATHS*sizeof(float)));
        testCUDA(hipMalloc((void **)&d_output3,blocks * sizeof(float)));
        //start time
        hipEventRecord(start, 0);
        


        simulateOptionPriceMultipleBlockGPU<<<blocksPerGrid,threads>>>( d_optionPriceGPU3,  K,  r,  T, sigma,  N_PATHS,  d_randomData,  N_STEPS, S0, dt, sqrdt);
        hipError_t error3 = hipGetLastError();
        if (error3 != hipSuccess) {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error3));
            return -1;
        }



        reduce3<<<blocks,threads>>>(d_optionPriceGPU3,d_output3,N_PATHS);
        error3 = hipGetLastError();
        if (error3 != hipSuccess) {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error3));
            return -1;
        }


        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        times_for_simulations[j] = milliseconds;

        testCUDA(hipMemcpy(output3, d_output3, blocks * sizeof(float), hipMemcpyDeviceToHost));


        cout << endl;
        float sum = 0.0f;
        for(int i=0; i<blocks; i++){
            sum+=output3[i];
        }
        cout<< "result gpu cuda option price vanilla " << expf(-r*T)*sum/N_PATHS << endl;

        hipFree(d_optionPriceGPU3);
        hipFree(d_output3);
        free(output3);
      }
      fprintf(file, "%d, %f, %f, %f, %f, %f, %f\n", number_of_simulations[i], times_for_simulations[0], times_for_simulations[1], times_for_simulations[2], times_for_simulations[3], times_for_simulations[4], times_for_simulations[5]);
    }





//--------------------------------BLACK SCHOLES FORMULA ----------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------

    float callResult = 0.0f;
    black_scholes_CPU(callResult,S0, K, T, r,  sigma);
    cout << endl << "call Black Scholes : " << callResult << endl;






//-------------------------------BULLET OPTION WITH MULTIPLE BLOCKS ------------------------------------------------------------
//------------------------------------------------------------------------------------------------------------------------------

  float *d_simulated_payoff_bullet, *h_simulated_payoff_bullet, *h_output4, *d_output4;
  testCUDA(hipMalloc((void **)&d_simulated_payoff_bullet, N_PATHS * sizeof(float)));
  testCUDA(hipMalloc((void **)&d_output4, blocks * sizeof(float)));
  h_output4 = (float *)malloc(blocks * sizeof(float));
  h_simulated_payoff_bullet = (float *)malloc(N_PATHS * sizeof(float));

  simulateBulletOptionPriceMultipleBlockGPU<<<blocksPerGrid,threadsPerBlock>>>( d_simulated_payoff_bullet,  K,  r,  T, sigma,  N_PATHS,  d_randomData,  N_STEPS, S0, dt, sqrdt, B, P1, P2);
  hipError_t error4 = hipGetLastError();
  if (error4 != hipSuccess) {
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error4));
      return -1;
  }
  hipDeviceSynchronize();
  testCUDA(hipMemcpy(h_simulated_payoff_bullet, d_simulated_payoff_bullet, N_PATHS * sizeof(float), hipMemcpyDeviceToHost));

  reduce3<<<blocks,threads>>>(d_simulated_payoff_bullet,d_output4,N_PATHS);
  error4 = hipGetLastError();
  if (error4 != hipSuccess) {
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error4));
      return -1;
  }
  hipDeviceSynchronize();
  testCUDA(hipMemcpy(h_output4, d_output4, blocks * sizeof(float), hipMemcpyDeviceToHost));
  float sum4 = 0.0f;
  for(int i=0; i<blocks; i++){
      sum4+=h_output4[i];
  }
  cout<< "result gpu cuda computed bullet option " << expf(-r*T) * sum4/N_PATHS << endl;



  //-------------------------------BULLET OPTION WITH MULTIPLE BLOCKS AND SAVE PATHS------------------------------------------------------------
  //-------------------------------------------------------------------------------------------------------------------------------------------

  float *d_simulated_paths, *d_simulated_count, *h_simulated_paths, *h_simulated_count;
  testCUDA(hipMalloc((void **)&d_simulated_paths, N_PATHS * N_STEPS * sizeof(float)));
  testCUDA(hipMalloc((void **)&d_simulated_count, N_PATHS * N_STEPS * sizeof(float)));
  h_simulated_paths = (float *)malloc(N_PATHS * N_STEPS * sizeof(float));
  h_simulated_count = (float *)malloc(N_PATHS * N_STEPS * sizeof(float));

  simulateBulletOptionSavePrice<<<blocksPerGrid,threadsPerBlock>>>( d_simulated_paths, d_simulated_count,  K,  r,  T, sigma,  N_PATHS,  d_randomData,  N_STEPS, S0, dt, sqrdt, B, P1, P2);
  hipError_t error5 = hipGetLastError();
  if (error5 != hipSuccess) {
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error5));
      return -1;
  }
  hipDeviceSynchronize();
  testCUDA(hipMemcpy(h_simulated_paths, d_simulated_paths, N_PATHS * N_STEPS * sizeof(float), hipMemcpyDeviceToHost));
  testCUDA(hipMemcpy(h_simulated_count, d_simulated_count, N_PATHS * N_STEPS * sizeof(float), hipMemcpyDeviceToHost)); 

  
  hipFree(d_simulated_paths);
  hipFree(d_simulated_count);
  free(h_simulated_paths);
  free(h_simulated_count);
  hipFree(d_simulated_payoff_bullet);
  hipFree(d_output4);
  free(h_output4);
  hipFree(d_randomData);



	return 0;
}


