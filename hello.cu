#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include  "pricinghost.hpp"
#include <random>
#include <hiprand.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <stdio.h>
#include <math.h>

using namespace std;
namespace cg = cooperative_groups;

extern "C" bool isPow2(unsigned int x) { return ((x & (x - 1)) == 0); }

unsigned int nextPow2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

// Function that catches the error
void testCUDA(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("There is an error in file %s at line %d\n", file, line);
        exit(EXIT_FAILURE);
    }
}
// Has to be defined in the compilation in order to get the correct value of the
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__, __LINE__))


// Cumulative normal distribution function
float NP(float x) {
    float p = 0.2316419f;
    float b1 = 0.31938153f;
    float b2 = -0.356563782f;
    float b3 = 1.781477937f;
    float b4 = -1.821255978f;
    float b5 = 1.330274429f;
    float one_over_twopi = 0.39894228f;
    float t;

    if (x >= 0.0f)
    {
        t = 1.0f / (1.0f + p * x);
        return (1.0f - one_over_twopi * expf(-x * x / 2.0f) * t *
               (t * (t * (t * (t * b5 + b4) + b3) + b2) + b1));
    }
    else
    {
        t = 1.0f / (1.0f - p * x);
        return (one_over_twopi * expf(-x * x / 2.0f) * t *
               (t * (t * (t * (t * b5 + b4) + b3) + b2) + b1));
    }
}




__global__ void reduce3(float *g_idata, float *g_odata, unsigned int n) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  __shared__ float sdata[1024];

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

  float mySum = (i < n) ? g_idata[i] : 0;
  

  if (i + blockDim.x < n) mySum += g_idata[i + blockDim.x];
  cg::sync(cta);
  sdata[tid] = mySum;



  // do reduction in shared mem
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
        mySum = mySum + sdata[tid + s];
        sdata[tid] = mySum;

    }

    cg::sync(cta);

  }


  // write result for this block to global mem
  if (tid == 0){
    printf("mySum last %f , %d \n", mySum, blockIdx.x);

    g_odata[blockIdx.x] = mySum;

  } 
}



__global__ void reduce4(float *g_idata, float *g_odata, unsigned int n) {
  // Handle to thread block group
  const int blockSize = 1024;
  cg::thread_block cta = cg::this_thread_block();
  __shared__ float sdata[1024];


  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

  float mySum = (i < n) ? g_idata[i] : 0;

  if (i + 1024 < n) mySum += g_idata[i + 1024];

  sdata[tid] = mySum;
  cg::sync(cta);

  // do reduction in shared mem
  for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
    if (tid < s) {
      sdata[tid] = mySum = mySum + sdata[tid + s];
    }

    cg::sync(cta);
  }

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  if (cta.thread_rank() < 32) {
    // Fetch final intermediate sum from 2nd warp
    if (blockSize >= 64) mySum += sdata[tid + 32];
    // Reduce final warp using shuffle
    for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
      mySum += tile32.shfl_down(mySum, offset);
    }
  }

  // write result for this block to global mem
  if (cta.thread_rank() == 0) g_odata[blockIdx.x] = mySum;
}

__global__ void reduce5(float *g_idata, float *g_odata, unsigned int n) {
  // Handle to thread block group
  const int blockSize = 1024;
  cg::thread_block cta = cg::this_thread_block();
  __shared__ float sdata[blockSize];

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * (blockSize * 2) + threadIdx.x;

  float mySum = (i < n) ? g_idata[i] : 0;

  if (i + blockSize < n) mySum += g_idata[i + blockSize];

  sdata[tid] = mySum;
  cg::sync(cta);

  // do reduction in shared mem
  if ( (blockSize >= 1024) && (tid < 512)) {
    sdata[tid] = mySum = mySum + sdata[tid + 512];
  }
  cg::sync(cta);
 if ((blockSize >= 512) && (tid < 256)) {
    sdata[tid] = mySum = mySum + sdata[tid + 256];
  }

  cg::sync(cta);

  if ((blockSize >= 256) && (tid < 128)) {
    sdata[tid] = mySum = mySum + sdata[tid + 128];
  }

  cg::sync(cta);

  if ((blockSize >= 128) && (tid < 64)) {
    sdata[tid] = mySum = mySum + sdata[tid + 64];
  }


  cg::sync(cta);

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  if (cta.thread_rank() < 32) {
    // Fetch final intermediate sum from 2nd warp
    if (blockSize >= 64) mySum += sdata[tid + 32];
    // Reduce final warp using shuffle
    for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
      mySum += tile32.shfl_down(mySum, offset);
    }
  }

  // write result for this block to global mem
  if (cta.thread_rank() == 0) g_odata[blockIdx.x] = mySum;
}




__global__ void reduce6(float *g_idata, float *g_odata, unsigned int n, bool nIsPow2) {
  // Handle to thread block group
  const int blockSize = 1024;
  cg::thread_block cta = cg::this_thread_block();
  __shared__ float sdata[blockSize];
  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int gridSize = blockSize * gridDim.x;

  float mySum = 0;

  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  if (nIsPow2) {
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    gridSize = gridSize << 1;

    while (i < n) {
      mySum += g_idata[i];
      // ensure we don't read out of bounds -- this is optimized away for
      // powerOf2 sized arrays
      if ((i + blockSize) < n) {
        mySum += g_idata[i + blockSize];
      }
      i += gridSize;
    }
  } else {
    unsigned int i = blockIdx.x * blockSize + threadIdx.x;
    while (i < n) {
      mySum += g_idata[i];
      i += gridSize;
    }
  }

  // each thread puts its local sum into shared memory
  sdata[tid] = mySum;
  cg::sync(cta);

// do reduction in shared mem
  if ( (blockSize >= 1024) && (tid < 512)) {
    sdata[tid] = mySum = mySum + sdata[tid + 512];
  }
  cg::sync(cta);
 if ((blockSize >= 512) && (tid < 256)) {
    sdata[tid] = mySum = mySum + sdata[tid + 256];
  }

  cg::sync(cta);

  if ((blockSize >= 256) && (tid < 128)) {
    sdata[tid] = mySum = mySum + sdata[tid + 128];
  }

  cg::sync(cta);

  if ((blockSize >= 128) && (tid < 64)) {
    sdata[tid] = mySum = mySum + sdata[tid + 64];
  }
  cg::sync(cta);


  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  if (cta.thread_rank() < 32) {
    // Fetch final intermediate sum from 2nd warp
    if (blockSize >= 64) mySum += sdata[tid + 32];
    // Reduce final warp using shuffle
    for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
      mySum += tile32.shfl_down(mySum, offset);
    }
  }

  // write result for this block to global mem
  if (cta.thread_rank() == 0) g_odata[blockIdx.x] = mySum;
}




void black_scholes_CPU(float &call_price, float x0, float strike_price, float T, float risk_free_rate, float volatility )
{

    float sqrtT = sqrtf(T);
    float    d1 = (logf(x0 / strike_price) + (risk_free_rate + 0.5 * volatility * volatility) * T) / (volatility * sqrtT);
    float    d2 = d1 - volatility * sqrtT;
    float cnd_d1 = CND(d1);
    float cnd_d2 = CND(d2);

    call_price   = x0 * cnd_d1 - strike_price * exp(- risk_free_rate * T) * cnd_d2;
}



__device__ inline float cndGPU(float d)
{
    const float       A1 = 0.31938153f;
    const float       A2 = -0.356563782f;
    const float       A3 = 1.781477937f;
    const float       A4 = -1.821255978f;
    const float       A5 = 1.330274429f;
    const float RSQRT2PI = 0.39894228040143267793994605993438f;

    float
    K = __fdividef(1.0f, (1.0f + 0.2316419f * fabsf(d)));

    float
    cnd = RSQRT2PI * __expf(- 0.5f * d * d) *
          (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));

    if (d > 0)
        cnd = 1.0f - cnd;

    return cnd;
}


///////////////////////////////////////////////////////////////////////////////
// Black-Scholes formula for both call and put
///////////////////////////////////////////////////////////////////////////////
__device__ inline void BlackScholesBodyGPU(
    float &CallResult,
    float &PutResult,
    float S, //Stock price
    float X, //Option strike
    float T, //Option years
    float R, //Riskless rate
    float V  //Volatility rate
)
{
    float sqrtT, expRT;
    float d1, d2, CNDD1, CNDD2;

    sqrtT = __fdividef(1.0F, rsqrtf(T));
    d1 = __fdividef(__logf(S / X) + (R + 0.5f * V * V) * T, V * sqrtT);
    d2 = d1 - V * sqrtT;

    CNDD1 = cndGPU(d1);
    CNDD2 = cndGPU(d2);

    //Calculate Call and Put simultaneously
    expRT = __expf(- R * T);
    CallResult = S * CNDD1 - X * expRT * CNDD2;
    PutResult  = X * expRT * (1.0f - CNDD2) - S * (1.0f - CNDD1);
}


////////////////////////////////////////////////////////////////////////////////
//Process an array of optN options on GPU
////////////////////////////////////////////////////////////////////////////////
__launch_bounds__(128)
__global__ void BlackScholesGPU(
    float2 * __restrict d_CallResult,
    float2 * __restrict d_PutResult,
    float2 * __restrict d_StockPrice,
    float2 * __restrict d_OptionStrike,
    float2 * __restrict d_OptionYears,
    float Riskfree,
    float Volatility,
    int optN
)
{
    ////Thread index
    //const int      tid = blockDim.x * blockIdx.x + threadIdx.x;
    ////Total number of threads in execution grid
    //const int THREAD_N = blockDim.x * gridDim.x;

    const int opt = blockDim.x * blockIdx.x + threadIdx.x;

     // Calculating 2 options per thread to increase ILP (instruction level parallelism)
    if (opt < (optN / 2))
    {
        float callResult1, callResult2;
        float putResult1, putResult2;
        BlackScholesBodyGPU(
            callResult1,
            putResult1,
            d_StockPrice[opt].x,
            d_OptionStrike[opt].x,
            d_OptionYears[opt].x,
            Riskfree,
            Volatility
        );
        BlackScholesBodyGPU(
            callResult2,
            putResult2,
            d_StockPrice[opt].y,
            d_OptionStrike[opt].y,
            d_OptionYears[opt].y,
            Riskfree,
            Volatility
        );
        d_CallResult[opt] = make_float2(callResult1, callResult2);
        d_PutResult[opt] = make_float2(putResult1, putResult2);
	 }
}




void generateRandomArray(float *d_randomData, float *h_randomData, int N_PATHS, int N_STEPS, unsigned long long seed = 1234ULL){

    // create generator all fill array with generated values
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hiprandGenerateNormal(gen, d_randomData, N_PATHS * N_STEPS, 0.0, 1.0);
    cout << endl << "number generated" << endl;
    testCUDA(hipMemcpy(h_randomData, d_randomData, N_PATHS * N_STEPS * sizeof(float), hipMemcpyDeviceToHost));
    cout << "host copied" << endl;
    hiprandDestroyGenerator(gen);

}






__global__ void simulateOptionPriceGPU(float *d_optionPriceGPU, float K, float r, float T,float sigma, int N_PATHS, float *d_randomData, int N_STEPS, float S0, float dt, float sqrdt) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N_PATHS) {
        float St = S0;
        float G;
        for(int i = 0; i < N_STEPS; i++){
            G = d_randomData[idx*i];
            // cout << "G : " << G << endl;
            St *= exp((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
        }
        
        // // Calculate the payoff
        d_optionPriceGPU[idx] = max(St - K, 0.0f);


    }
}


__global__ void simulateOptionPriceOneBlockGPUSumReduce(float *d_optionPriceGPU, float K, float r, float T,float sigma, int N_PATHS, float *d_randomData, int N_STEPS, float S0, float dt, float sqrdt, float *output) {
    int stride = blockDim.x;
    int idx = threadIdx.x;
    int tid = threadIdx.x;

    // Shared memory for the block
    __shared__ float sdata[1024];
    float sum = 0.0f;
    
    if(idx < N_PATHS) {
        sdata[tid] = 0.0f;

        while(idx < N_PATHS){
            float St = S0;
            float G;
            for(int i = 0; i < N_STEPS; i++){
                G = d_randomData[idx*N_STEPS + i];
                // cout << "G : " << G << endl;
                St *= exp((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
            }
            sum += max(St - K,0.0f);
            idx += stride;
        }
    // Load input into shared memory
        sdata[tid] = (tid < N_PATHS) ? sum : 0;

        __syncthreads();

        // Perform reduction in shared memory
        for (unsigned int s = 1024 / 2; s > 0; s >>= 1) {
            if (tid < s && (tid + s) < N_PATHS) {
                sdata[tid] += sdata[tid + s];
            }
            __syncthreads();
        }

        // Write result for this block to output
        if (tid == 0){
            output[0] = sdata[0];
            }  
        }  
}


__global__ void simulateOptionPriceMultipleBlockGPU(float *d_simulated_payoff, float K, float r, float T,float sigma, int N_PATHS, float *d_randomData, int N_STEPS, float S0, float dt, float sqrdt) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx < N_PATHS) {
            float St = S0;
            float G;
            for(int i = 0; i < N_STEPS; i++){
                G = d_randomData[idx*N_STEPS + i];
                St *= expf((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
            }
            d_simulated_payoff[idx] = max(St - K,0.0f);
            printf("St : %f \n", St);
        }
    }

__global__ void simulateBulletOptionPriceMultipleBlockGPU(float *d_simulated_payoff, float K, float r, float T,float sigma, int N_PATHS, float *d_randomData, int N_STEPS, float S0, float dt, float sqrdt, float B, float P1, float P2) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(idx < N_PATHS) {
    printf("idx : %d \n", idx);
    int count = 0;
    float St = S0;
    float G;
    for(int i = 0; i < N_STEPS; i++){
        G = d_randomData[idx*N_STEPS + i];
        St *= expf((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
    }
    printf("St : %f \n", St);
    printf("count : %d \n", count);
    if((count >= P1) && (count <= P2)){
      d_simulated_payoff[idx] = max(St - K,0.0f);
    } else {
      d_simulated_payoff[idx] = 0.0f;
    }
    printf("payoff : %f \n", d_simulated_payoff[idx]);
  }
}


void getDeviceProperty(){

    int count;
    hipDeviceProp_t prop;
    hipGetDeviceCount(&count);
    printf("The number of devices available is %i GPUs \n", count);
    hipGetDeviceProperties(&prop, count-1);
    printf("Name: %s\n", prop.name);
    printf("Global memory size in bytes: %ld\n", prop.totalGlobalMem);
    printf("Shared memory size per block: %ld\n", prop.sharedMemPerBlock);
    printf("Number of registers per block: %d\n", prop.regsPerBlock);
    printf("Number of threads in a warp: %d\n", prop.warpSize);
    printf("Maximum number of threads that can be launched per block: %d\n", prop.maxThreadsPerBlock);
    printf("Maximum number of threads that can be launched: %d x %d x %d\n",
           prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Maximum grid size: %d x %d x %d\n",
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Total constant memory: %ld\n", prop.totalConstMem);
    printf("Major compute capability: %d\n", prop.major);
    printf("Minor compute capability: %d\n", prop.minor);
    printf("Clock rate: %d\n", prop.clockRate);
    printf("Maximum 1D texture memory: %d\n", prop.maxTexture1D);
    printf("Could we overlap? %d\n", prop.deviceOverlap);
    printf("Number of multiprocessors: %d\n", prop.multiProcessorCount);
    printf("Is there a limit for kernel execution? %d\n", prop.kernelExecTimeoutEnabled);
    printf("Is my GPU a chipsest? %d\n", prop.integrated);
    printf("Can we map the host memory? %d\n", prop.canMapHostMemory);
    printf("Can we launch concurrent kernels? %d\n", prop.concurrentKernels);
    printf("Do we have ECC memory? %d\n", prop.ECCEnabled);

}

void simulateOptionPriceCPU(float *optionPriceCPU, int N_PATHS, int N_STEPS, float * h_randomData, float S0, float sigma, float sqrdt, float r, float K, float dt, float *simulated_paths_cpu){
    float G;
    float countt = 0.0f;
    for(int i=0; i<N_PATHS;i++){
        float St = S0;
        for(int j=0; j<N_STEPS; j++){
            G = h_randomData[i*N_STEPS + j];
            St *= expf((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
            
        }

        simulated_paths_cpu[i] = max(St - K, 0.0f);
        // cout << "cpu : " <<  St << endl;
        countt += max(St - K, 0.0f);
    }
    *optionPriceCPU = countt/N_PATHS;
}






int main(void) {



// declare variables and constants
    unsigned int N_PATHS = 10;
    const size_t N_STEPS = 100;
    const float T = 1.0f;
    const float K = 100.0f;
    const float B = 110.0f;
    const float S0 = 100.0f;
    const float sigma = 0.2f;
    const float r = 0.05f;
    float dt = float(T)/float(N_STEPS);
    float sqrdt = sqrt(dt);
    vector<float> s(N_PATHS);
    int threadsPerBlock = 1024;
    unsigned int maxThreads = 1024;
    int P1 = 10;
    int P2 = 50;

    getDeviceProperty();

    int blocksPerGrid = (N_PATHS + threadsPerBlock - 1) / threadsPerBlock;

    cout << "number of paths : " << N_PATHS << endl;
    cout << "number of steps : " << N_STEPS << endl;


    float *d_randomData, *h_randomData, *simulated_paths_cpu;
    testCUDA(hipMalloc(&d_randomData, N_PATHS * N_STEPS * sizeof(float)));
    h_randomData = (float *)malloc(N_PATHS * N_STEPS*sizeof(float));
    simulated_paths_cpu = (float *)malloc(N_PATHS *sizeof(float));
    generateRandomArray(d_randomData, h_randomData, N_PATHS, N_STEPS);


    cout << "random  " << h_randomData[0] << endl;

    float optionPriceCPU = 0.0f;
    simulateOptionPriceCPU(&optionPriceCPU,  N_PATHS,  N_STEPS,  h_randomData,  S0,  sigma,  sqrdt,  r, K, dt,simulated_paths_cpu);

    cout << endl;
    cout << "Average CPU : " << optionPriceCPU << endl << endl;



//--------------------------------GPU WITH ONE BLOCK ----------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------

    float *h_optionPriceGPU, *output;
    h_optionPriceGPU = (float *)malloc(N_PATHS * sizeof(float));
    output = (float *)malloc(sizeof(float));
    float *d_optionPriceGPU, *d_output;

    testCUDA(hipMalloc((void **)&d_optionPriceGPU,N_PATHS*sizeof(float)));
    testCUDA(hipMalloc((void **)&d_output,sizeof(float)));

    simulateOptionPriceOneBlockGPUSumReduce<<<1, 1024>>>( d_optionPriceGPU,  K,  r,  T, sigma,  N_PATHS,  d_randomData,  N_STEPS, S0, dt, sqrdt, d_output);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return -1;
    }
    hipMemcpy(h_optionPriceGPU, d_optionPriceGPU, N_PATHS * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(output, d_output, sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    cout << endl;

    cout << "Average GPU one block : " << output[0]/N_PATHS << endl ;

    hipFree(d_optionPriceGPU);
    hipFree(d_output);
    free(h_optionPriceGPU);
    free(output);


//--------------------------------GPU WITH MULTIPLE BLOCK ----------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------


    int threads = (N_PATHS < maxThreads * 2) ? nextPow2((N_PATHS + 1) / 2) : maxThreads;
    int blocks = (N_PATHS + (threads * 2 - 1)) / (threads * 2);


    float *output3, *d_optionPriceGPU3, *d_output3;
    output3 = (float *)malloc(blocks * sizeof(float));

    testCUDA(hipMalloc((void **)&d_optionPriceGPU3,N_PATHS*sizeof(float)));
    testCUDA(hipMalloc((void **)&d_output3,blocks * sizeof(float)));

    cout << "number of blocks : " << blocksPerGrid << endl;
    cout << "number of threads : " << threadsPerBlock << endl;

    simulateOptionPriceMultipleBlockGPU<<<blocksPerGrid,threadsPerBlock>>>( d_optionPriceGPU3,  K,  r,  T, sigma,  N_PATHS,  d_randomData,  N_STEPS, S0, dt, sqrdt);
    hipError_t error3 = hipGetLastError();
    if (error3 != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error3));
        return -1;
    }

    cout << "number of blocks" << blocks << endl;
    cout << "number of threads" << threads << endl;

    reduce3<<<blocks,threads>>>(d_optionPriceGPU3,d_output3,N_PATHS);
    error3 = hipGetLastError();
    if (error3 != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error3));
        return -1;
    }


    hipDeviceSynchronize();

    testCUDA(hipMemcpy(output3, d_output3, blocks * sizeof(float), hipMemcpyDeviceToHost));


    cout << endl;
    float sum = 0.0f;
    for(int i=0; i<blocks; i++){
        sum+=output3[i];
    }
    cout<< "result gpu cuda computed " << sum/N_PATHS << endl;

    hipFree(d_optionPriceGPU3);
    hipFree(d_output3);
    free(output3);





//--------------------------------BLACK SCHOLES FORMULA ----------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------

    float callResult = 0.0f;
    black_scholes_CPU(callResult,S0, K, T, r,  sigma);
    cout << "call BS : " << callResult << endl;



    hipFree(d_randomData);


//-------------------------------BULLET OPTION WITH MULTIPLE BLOCKS ------------------------------------------------------------
//------------------------------------------------------------------------------------------------------------------------------

  float *d_simulated_payoff_bullet, *h_simulated_payoff_bullet, *h_output4, *d_output4;
  testCUDA(hipMalloc((void **)&d_simulated_payoff_bullet, N_PATHS * sizeof(float)));
  testCUDA(hipMalloc((void **)&d_output4, blocks * sizeof(float)));
  h_output4 = (float *)malloc(blocks * sizeof(float));
  h_simulated_payoff_bullet = (float *)malloc(N_PATHS * sizeof(float));

  simulateBulletOptionPriceMultipleBlockGPU<<<blocksPerGrid,threadsPerBlock>>>( d_simulated_payoff_bullet,  K,  r,  T, sigma,  N_PATHS,  d_randomData,  N_STEPS, S0, dt, sqrdt, B, P1, P2);
  hipError_t error4 = hipGetLastError();
  if (error4 != hipSuccess) {
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error4));
      return -1;
  }
  hipDeviceSynchronize();
  testCUDA(hipMemcpy(h_simulated_payoff_bullet, d_simulated_payoff_bullet, N_PATHS * sizeof(float), hipMemcpyDeviceToHost));
  for(int i=0; i<N_PATHS; i++){
      cout << "simulated payoff bullet : " << h_simulated_payoff_bullet[i] << endl;
  }

  reduce3<<<blocks,threads>>>(d_simulated_payoff_bullet,d_output4,N_PATHS);
  error4 = hipGetLastError();
  if (error4 != hipSuccess) {
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error4));
      return -1;
  }
  hipDeviceSynchronize();
  testCUDA(hipMemcpy(h_output4, d_output4, blocks * sizeof(float), hipMemcpyDeviceToHost));
  float sum4 = 0.0f;
  for(int i=0; i<blocks; i++){
      sum4+=h_output4[i];
  }
  cout<< "result gpu cuda computed bullet option " << sum4/N_PATHS << endl;
  
  hipFree(d_simulated_payoff_bullet);
  hipFree(d_output4);
  free(h_output4);




	return 0;
}


