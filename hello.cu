#include "hip/hip_runtime.h"
#include <iostream>
// #include <format>
// #include <functional>
#include <hip/hip_runtime.h>

#include "trajectories.hpp"
#include "common.hpp"
#include "Xoshiro.hpp"
#include  "pricinghost.hpp"
#include <random>
#include <hiprand.h>

// Function that catches the error
void testCUDA(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("There is an error in file %s at line %d\n", file, line);
        exit(EXIT_FAILURE);
    }
}


__global__ void setValuesKernel(float *arr, float value, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        arr[idx] = value;
    }
}




// Has to be defined in the compilation in order to get the correct value of the
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__, __LINE__))


using namespace std;

__global__ void simulateOptionPrice(float *d_optionPriceGPU, float K, float r, float T,float sigma, int N_PATHS, float *d_randomData, int N_STEPS, float S0, float dt, float sqrdt) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N_PATHS) {
        // float St = S0;
        // float G;
        // for(int i = 0; i < N_STEPS; i++){
        //     G = d_randomData[idx*i];
        //     // cout << "G : " << G << endl;
        //     St *= exp((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
        // }
        
        // // Calculate the payoff
        d_optionPriceGPU[idx] = 1.0f;
    }
}


void addVect(int *a, int *b, int *c, int length)
{

	int i;

	for (i = 0; i < length; i++)
	{
		c[i] = a[i] + b[i];
	}
}

__global__ void cudaAdd(int *d_a, int *d_b, int *d_c, int length)
{
	int indice = threadIdx.x + blockIdx.x * blockDim.x;
	if (indice < length)
	{
		d_c[indice] = d_a[indice] + d_b[indice];
	}
}



int main(void) {

// declare variables and constants
    const size_t N_PATHS = 10;
    const size_t N_STEPS = 5;
    const size_t N_NORMALS = N_PATHS*N_STEPS;
    const float T = 1.0f;
    const float K = 100.0f;
    const float B = 95.0f;
    const float S0 = 100.0f;
    const float sigma = 0.2f;
    const float mu = 0.1f;
    const float r = 0.05f;
    float dt = float(T)/float(N_STEPS);
    float sqrdt = sqrt(dt);

    vector<float> s(N_PATHS);

    float step = 1.0f / N_STEPS;
    float G = 0.0f;
    std::default_random_engine generator;
    std::normal_distribution<double> distribution(0.0, 1.0);

    cout << "step : " << step << endl;
    G = distribution(generator);
    // cout << "G : " << G;






    // generate random numbers using hiprand

    //allocate array filled with random values 
    float *d_randomData;
    testCUDA(hipMalloc(&d_randomData, N_PATHS * N_STEPS * sizeof(float)));

    // create generator all fill array with generated values
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    hiprandGenerateNormal(gen, d_randomData, N_PATHS * N_STEPS, 0.0, 1.0);

    cout << "number generated";




    float h_randomData[N_PATHS * N_STEPS];
    testCUDA(hipMemcpy(h_randomData, d_randomData, N_PATHS * N_STEPS * sizeof(float), hipMemcpyDeviceToHost));

    cout << "host copied" << endl;
    cout << h_randomData[0];

    // for(int i = 0; i < N_PATHS * N_STEPS; i++) {
    //     cout << "random  : " << h_randomData[i] << endl;
    // }

    float count = 0.0f;
    for(int i=0; i<N_PATHS;i++){
        float St = S0;
        for(int j=0; j<N_STEPS; j++){
            G = h_randomData[i*j];
            // cout << "G : " << G << endl;
            St *= exp((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
            
        }
        // cout << "S before assigning " << St << endl;
        s[i] = St;
        count += St;
        cout << "St : " << St << endl;
        // cout << "S " << St << endl;
        // cout << i << endl;
    }
    cout << "paths calculated" << endl;
    cout << "mean paths : " << count/N_PATHS << endl;



    // simulateOptionPrice<<<1, N_PATHS>>>( d_optionPriceGPU,  K,  r,  T, sigma,  N_PATHS,  d_randomData,  N_STEPS, S0, dt, sqrdt);
    // hipDeviceSynchronize();


    // float *a;
    // a = (float *)malloc(N_PATHS * sizeof(float));
    // float *d_a;
    // testCUDA(hipMalloc((void **)&d_a,N_PATHS*sizeof(float)));

    // simulateOptionPrice<<<1, N_PATHS>>>( d_a,  K,  r,  T, sigma,  N_PATHS,  d_randomData,  N_STEPS, S0, dt, sqrdt);
    // hipDeviceSynchronize();

    // hipMemcpy(a, d_a, N_PATHS * sizeof(float), hipMemcpyDeviceToHost);

    // for(int i = 0; i<N_PATHS; i++){
    //     cout << "GPU St : " << a[i] << endl;
    // }

// Variables definition
	int *a, *b, *c, *c_cuda;
	int *d_a, *d_b, *d_c;
	int i;

	// Length for the size of arrays
	int length = 20;



	// Memory allocation of arrays
	a = (int *)malloc(length * sizeof(int));
	b = (int *)malloc(length * sizeof(int));
	c = (int *)malloc(length * sizeof(int));
	c_cuda = (int *)malloc(length * sizeof(int));

	// device memory allocation

	hipMalloc((void **)&d_a, length * sizeof(int));
	hipMalloc((void **)&d_b, length * sizeof(int));
	hipMalloc((void **)&d_c, length * sizeof(int));

	
	// Setting value
	for (i = 0; i < length; i++)
	{
		a[i] = i;
		b[i] = 9 * i;
	}

	hipMemcpy(d_a, a, length * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, length * sizeof(int), hipMemcpyHostToDevice);



	// Executing the addition
	addVect(a, b, c, length);

	Tim.add(); // CPU timer instructions
	int nbBlock = (length + 1024 - 1) / 1024;
	int nbthread = 1024;

	cudaAdd<<<nbBlock, nbthread>>>(d_a, d_b, d_c, length);

	hipEventRecord(end);

	hipMemcpy(c_cuda, d_c, length * sizeof(int), hipMemcpyDeviceToHost);
	hipEventSynchronize();
	// Displaying the results to check the correctness
	for (int i = 0; i < length; i++)
	{

		printf("%f",c_cuda[i]);

	}






	// Freeing the memory
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;

    // float *d_optionPriceGU;
    // testCUDA(hipMalloc(&d_optionPriceGPU,N_PATHS*sizeof(float)));

    // int blockSize = 256; // You can adjust this based on your GPU's capability
    // int numBlocks = (N_PATHS + blockSize - 1) / blockSize;

    // cout << "nb block" << numBlocks << endl;

    // initializeArray<<<numBlocks, blockSize>>>(d_optionPriceGPU, N_PATHS, 6.0f);
    // hipError_t err = hipGetLastError();
    // if (err != hipSuccess) {
    //     cout << "CUDA Error: " << hipGetErrorString(err) << endl;
    // }
    // testCUDA(hipDeviceSynchronize());
    // float *h_optionPriceGPU = new float[N_PATHS];
    // testCUDA(hipMemcpy(h_optionPriceGPU, d_optionPriceGPU,N_PATHS*sizeof(float),hipMemcpyDeviceToHost));

    // for(int i = 0; i<N_PATHS; i++){
    //     cout << "GPU St : " << h_optionPriceGPU[i] << endl;
    // }
    // cout << "mean paths GPU : " << mean_priceGPU/N_PATHS << endl;


    testCUDA(hipFree(d_randomData));
    hiprandDestroyGenerator(gen);

	return 0;
}


