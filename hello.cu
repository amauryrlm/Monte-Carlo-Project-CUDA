#include "hip/hip_runtime.h"
#include <iostream>
// #include <format>
// #include <functional>
#include <hip/hip_runtime.h>

#include "trajectories.hpp"
#include "common.hpp"
#include "Xoshiro.hpp"
#include  "pricinghost.hpp"
#include <random>
#include <hiprand.h>

// Function that catches the error
void testCUDA(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("There is an error in file %s at line %d\n", file, line);
        exit(EXIT_FAILURE);
    }
}

// Has to be defined in the compilation in order to get the correct value of the
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__, __LINE__))


using namespace std;

__global__ void simulateOptionPrice(float *d_optionPriceGPU, float K, float r, float T,float sigma, int N_PATHS, float *d_randomData, int N_STEPS, float S0, float dt, float sqrdt) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N_PATHS) {
        // float St = S0;
        // float G;
        // for(int i = 0; i < N_STEPS; i++){
        //     G = d_randomData[idx*i];
        //     // cout << "G : " << G << endl;
        //     St *= exp((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
        // }
        
        // // Calculate the payoff
        d_optionPriceGPU[idx] = 1.0f;
    }
}



int main(void) {

// declare variables and constants
    const size_t N_PATHS = 10;
    const size_t N_STEPS = 5;
    const size_t N_NORMALS = N_PATHS*N_STEPS;
    const float T = 1.0f;
    const float K = 100.0f;
    const float B = 95.0f;
    const float S0 = 100.0f;
    const float sigma = 0.2f;
    const float mu = 0.1f;
    const float r = 0.05f;
    float dt = float(T)/float(N_STEPS);
    float sqrdt = sqrt(dt);

    vector<float> s(N_PATHS);

    float step = 1.0f / N_STEPS;
    float G = 0.0f;
    std::default_random_engine generator;
    std::normal_distribution<double> distribution(0.0, 1.0);

    cout << "step : " << step << endl;
    G = distribution(generator);
    // cout << "G : " << G;






    // generate random numbers using hiprand

    //allocate array filled with random values 
    float *d_randomData;
    testCUDA(hipMalloc(&d_randomData, N_PATHS * N_STEPS * sizeof(float)));

    // create generator all fill array with generated values
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    hiprandGenerateNormal(gen, d_randomData, N_PATHS * N_STEPS, 0.0, 1.0);

    cout << "number generated";


    float h_randomData[N_PATHS * N_STEPS];
    testCUDA(hipMemcpy(h_randomData, d_randomData, N_PATHS * N_STEPS * sizeof(float), hipMemcpyDeviceToHost));

    cout << "host copied" << endl;
    cout << h_randomData[0];

    // for(int i = 0; i < N_PATHS * N_STEPS; i++) {
    //     cout << "random  : " << h_randomData[i] << endl;
    // }

    float count = 0.0f;
    for(int i=0; i<N_PATHS;i++){
        float St = S0;
        for(int j=0; j<N_STEPS; j++){
            G = h_randomData[i*j];
            // cout << "G : " << G << endl;
            St *= exp((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
            
        }
        // cout << "S before assigning " << St << endl;
        s[i] = St;
        count += St;
        cout << "St : " << St << endl;
        // cout << "S " << St << endl;
        // cout << i << endl;
    }
    cout << "paths calculated" << endl;
    cout << "mean paths : " << count/N_PATHS << endl;




    float *d_optionPriceGPU;
    testCUDA(hipMalloc(&d_optionPriceGPU,N_PATHS*sizeof(float)));
    testCUDA(hipMemset(d_optionPriceGPU, 6.0f, N_PATHS * sizeof(float)));

    simulateOptionPrice<<<1, N_PATHS>>>( d_optionPriceGPU,  K,  r,  T, sigma,  N_PATHS,  d_randomData,  N_STEPS, S0, dt, sqrdt);
    hipDeviceSynchronize();
    float *h_optionPriceGPU = new float[N_PATHS];
    testCUDA(hipMemcpy(h_optionPriceGPU, d_optionPriceGPU,N_PATHS*sizeof(float),hipMemcpyDeviceToHost));
    float mean_priceGPU = 0.0f;

    for(int i = 0; i<N_PATHS; i++){
        mean_priceGPU += h_optionPriceGPU[i];
        cout << "GPU St : " << h_optionPriceGPU[i] << endl;
    }
    cout << "mean paths GPU : " << mean_priceGPU/N_PATHS << endl;


    testCUDA(hipFree(d_randomData));
    hiprandDestroyGenerator(gen);

	return 0;
}

