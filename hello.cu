#include "hip/hip_runtime.h"
#include <iostream>
// #include <format>
// #include <functional>
#include <hip/hip_runtime.h>

#include "trajectories.hpp"
#include "common.hpp"
#include "Xoshiro.hpp"
#include  "pricinghost.hpp"
#include <random>
#include <hiprand.h>

__global__ void addKernel(int *c, const int *a, const int *b, int size) {
    int i = threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

// Function that catches the error
void testCUDA(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("There is an error in file %s at line %d\n", file, line);
        exit(EXIT_FAILURE);
    }
}

void addWithCuda(int *c, const int *a, const int *b, int size) {
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    // Allocate GPU buffers for three vectors (two input, one output).
    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b, size);
    // After kernel launch
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Wait for kernel to finish and check for any errors.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        goto Error;
    }

    Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "AddWithCuda failed!\n");
        return;
    }

}








// Has to be defined in the compilation in order to get the correct value of the
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__, __LINE__))


using namespace std;

__global__ void simulateOptionPrice(float *d_optionPriceGPU, float K, float r, float T,float sigma, int N_PATHS, float *d_randomData, int N_STEPS, float S0, float dt, float sqrdt) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N_PATHS) {
        // float St = S0;
        // float G;
        // for(int i = 0; i < N_STEPS; i++){
        //     G = d_randomData[idx*i];
        //     // cout << "G : " << G << endl;
        //     St *= exp((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
        // }
        
        // // Calculate the payoff
        d_optionPriceGPU[idx] = 1.0f;
    }
}





int main(void) {

    int count;
    hipDeviceProp_t prop;
    
    hipGetDeviceCount(&count);
    printf("The number of devices available is %i GPUs \n", count);
    hipGetDeviceProperties(&prop, count-1);
    printf("Name: %s\n", prop.name);
    printf("Global memory size in bytes: %ld\n", prop.totalGlobalMem);
    printf("Shared memory size per block: %ld\n", prop.sharedMemPerBlock);
    printf("Number of registers per block: %d\n", prop.regsPerBlock);
    printf("Number of threads in a warp: %d\n", prop.warpSize);
    printf("Maximum number of threads that can be launched per block: %d\n", prop.maxThreadsPerBlock);
    printf("Maximum number of threads that can be launched: %d x %d x %d\n",
           prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Maximum grid size: %d x %d x %d\n",
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Total constant memory: %ld\n", prop.totalConstMem);
    printf("Major compute capability: %d\n", prop.major);
    printf("Minor compute capability: %d\n", prop.minor);
    printf("Clock rate: %d\n", prop.clockRate);
    printf("Maximum 1D texture memory: %d\n", prop.maxTexture1D);
    printf("Could we overlap? %d\n", prop.deviceOverlap);
    printf("Number of multiprocessors: %d\n", prop.multiProcessorCount);
    printf("Is there a limit for kernel execution? %d\n", prop.kernelExecTimeoutEnabled);
    printf("Is my GPU a chipsest? %d\n", prop.integrated);
    printf("Can we map the host memory? %d\n", prop.canMapHostMemory);
    printf("Can we launch concurrent kernels? %d\n", prop.concurrentKernels);
    printf("Do we have ECC memory? %d\n", prop.ECCEnabled);

// declare variables and constants
    const size_t N_PATHS = 10;
    const size_t N_STEPS = 5;
    const size_t N_NORMALS = N_PATHS*N_STEPS;
    const float T = 1.0f;
    const float K = 100.0f;
    const float B = 95.0f;
    const float S0 = 100.0f;
    const float sigma = 0.2f;
    const float mu = 0.1f;
    const float r = 0.05f;
    float dt = float(T)/float(N_STEPS);
    float sqrdt = sqrt(dt);

    vector<float> s(N_PATHS);

    float step = 1.0f / N_STEPS;
    float G = 0.0f;
    std::default_random_engine generator;
    std::normal_distribution<double> distribution(0.0, 1.0);

    cout << "step : " << step << endl;
    G = distribution(generator);
    // cout << "G : " << G;






    // generate random numbers using hiprand

    //allocate array filled with random values 
    float *d_randomData;
    testCUDA(hipMalloc(&d_randomData, N_PATHS * N_STEPS * sizeof(float)));

    // create generator all fill array with generated values
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    hiprandGenerateNormal(gen, d_randomData, N_PATHS * N_STEPS, 0.0, 1.0);

    cout << "number generated";




    float h_randomData[N_PATHS * N_STEPS];
    testCUDA(hipMemcpy(h_randomData, d_randomData, N_PATHS * N_STEPS * sizeof(float), hipMemcpyDeviceToHost));

    cout << "host copied" << endl;
    cout << h_randomData[0];

    // for(int i = 0; i < N_PATHS * N_STEPS; i++) {
    //     cout << "random  : " << h_randomData[i] << endl;
    // }

    float countt = 0.0f;
    for(int i=0; i<N_PATHS;i++){
        float St = S0;
        for(int j=0; j<N_STEPS; j++){
            G = h_randomData[i*j];
            // cout << "G : " << G << endl;
            St *= exp((r - (sigma*sigma)/2)*dt + sigma * sqrdt * G);
            
        }
        // cout << "S before assigning " << St << endl;
        s[i] = St;
        countt += St;
        cout << "St : " << St << endl;
        // cout << "S " << St << endl;
        // cout << i << endl;
    }
    cout << "paths calculated" << endl;
    cout << "mean paths : " << countt/N_PATHS << endl;

    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    addWithCuda(c, a, b, arraySize);

    // Print the result.
    printf("Result:\n");
    for(int i = 0; i < arraySize; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }




    // simulateOptionPrice<<<1, N_PATHS>>>( d_optionPriceGPU,  K,  r,  T, sigma,  N_PATHS,  d_randomData,  N_STEPS, S0, dt, sqrdt);
    // hipDeviceSynchronize();




    // float *a;
    // a = (float *)malloc(N_PATHS * sizeof(float));
    // float *d_a;
    // testCUDA(hipMalloc((void **)&d_a,N_PATHS*sizeof(float)));

    // simulateOptionPrice<<<1, N_PATHS>>>( d_a,  K,  r,  T, sigma,  N_PATHS,  d_randomData,  N_STEPS, S0, dt, sqrdt);
    // hipDeviceSynchronize();

    // hipMemcpy(a, d_a, N_PATHS * sizeof(float), hipMemcpyDeviceToHost);
    // hipDeviceSynchronize();
    // for(int i = 0; i<N_PATHS; i++){
    //     cout << "GPU St : " << a[i] << endl;
    // }



    // float *d_optionPriceGU;
    // testCUDA(hipMalloc(&d_optionPriceGPU,N_PATHS*sizeof(float)));

    // int blockSize = 256; // You can adjust this based on your GPU's capability
    // int numBlocks = (N_PATHS + blockSize - 1) / blockSize;

    // cout << "nb block" << numBlocks << endl;

    // initializeArray<<<numBlocks, blockSize>>>(d_optionPriceGPU, N_PATHS, 6.0f);
    // hipError_t err = hipGetLastError();
    // if (err != hipSuccess) {
    //     cout << "CUDA Error: " << hipGetErrorString(err) << endl;
    // }
    // testCUDA(hipDeviceSynchronize());
    // float *h_optionPriceGPU = new float[N_PATHS];
    // testCUDA(hipMemcpy(h_optionPriceGPU, d_optionPriceGPU,N_PATHS*sizeof(float),hipMemcpyDeviceToHost));

    // for(int i = 0; i<N_PATHS; i++){
    //     cout << "GPU St : " << h_optionPriceGPU[i] << endl;
    // }
    // cout << "mean paths GPU : " << mean_priceGPU/N_PATHS << endl;


    testCUDA(hipFree(d_randomData));
    hiprandDestroyGenerator(gen);

	return 0;
}


